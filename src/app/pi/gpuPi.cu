#include "hip/hip_runtime.h"
#include "gpuPi.h"

#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#include "cuda/mycuda.h"


static CUDA_GLOBAL void computePi(unsigned int width, unsigned int timeSeed, unsigned int* result) {
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= width) return;
    hiprandState s;
    hiprand_init(timeSeed, index, 0, &s);
    float x = hiprand_uniform(&s) * 2 - 1;
    float y = hiprand_uniform(&s) * 2 - 1;
    float dist = x*x + y*y;

    const bool isInCircle = (dist <= 1.0f);
    const bool resIndex = static_cast<int>(isInCircle);
    atomicAdd(result + resIndex, 1);
}

CUDA_HOST double GpuPi::compute() {
    unsigned int seed = std::chrono::system_clock::now().time_since_epoch().count();

    unsigned int hostResult[2] = {0};
    unsigned int* devResult;
    hipMalloc(&devResult, sizeof(hostResult));
    hipMemcpy(devResult, hostResult, sizeof(hostResult), hipMemcpyHostToDevice);

    auto bloc = 1024;
    int grid = _iter / bloc + 1;
    computePi<<<grid, bloc>>>(_iter, seed, devResult);

    hipDeviceSynchronize();

    hipMemcpy(hostResult, devResult, sizeof(hostResult), hipMemcpyDeviceToHost);
    hipFree(devResult);

    unsigned int squareCount = hostResult[0];
    unsigned int circleCount = hostResult[1];
    const auto ratio  = static_cast<double>(circleCount) / (squareCount + circleCount);
    return ratio * 4;
}
